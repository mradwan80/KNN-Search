#include "hip/hip_runtime.h"
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<thrust/device_ptr.h>
#include<thrust/sort.h>
#include<thrust/sequence.h>
#include<thrust/gather.h>
#include<thrust/count.h>
#include <thrust/execution_policy.h>
#include<thrust/copy.h>
#include "DDS.h"

__global__ void CopyCountsKernel(int qnum, int len, float searchRad, int globalW, int globalH, float* pvmMat, float* vpos, int* xfcount, int* xfoffset, int* FragVertex, bool* pixelIn, int* sncount)
{
	int qspxl = blockIdx.x * blockDim.x + threadIdx.x;

	if (qspxl < qnum * len * len)
	{
		int q = qspxl / (len * len);
		int spxl = qspxl % (len * len);
		int sx = spxl % len;
		int sy = spxl / len;

		if (!pixelIn[spxl])
			return;

		//get vertex//
		float qx = vpos[3 * q + 0];
		float qy = vpos[3 * q + 1];
		float qz = vpos[3 * q + 2];
		float qw = 1.0;

		float posXpvm = pvmMat[0] * qx + pvmMat[4] * qy + pvmMat[8] * qz + pvmMat[12] * qw;
		float posYpvm = pvmMat[1] * qx + pvmMat[5] * qy + pvmMat[9] * qz + pvmMat[13] * qw;
		float posZpvm = pvmMat[2] * qx + pvmMat[6] * qy + pvmMat[10] * qz + pvmMat[14] * qw;
		float posWpvm = pvmMat[3] * qx + pvmMat[7] * qy + pvmMat[11] * qz + pvmMat[15] * qw;

		//exact pixel of q//
		int qxscreen = (int)(((posXpvm / posWpvm) / 2 + 0.5) * globalW);
		int qyscreen = (int)(((posYpvm / posWpvm) / 2 + 0.5) * globalH);

		//pixel in the square, based on thread//
		int xscreen = (qxscreen - len/2) + sx;
		int yscreen = (qyscreen - len/2) + sy;
		int pxl = xscreen + yscreen * globalW;

		if (xscreen<0 || xscreen>globalW - 1 || yscreen<0 || yscreen>globalH - 1)
			return;

		if (xfcount[pxl] == 0)
			return;

		int offset = xfoffset[pxl];
		int pcount = 0;
		for (int f = 0; f < xfcount[pxl]; f++)
		{
			int v = FragVertex[f + offset]; //get vertex//
			
			//get pos
			float x = vpos[3 * v + 0];
			float y = vpos[3 * v + 1];
			float z = vpos[3 * v + 2];
			float w = 1.0;

			float dist = (x - qx) * (x - qx) + (y - qy) * (y - qy) + (z - qz) * (z - qz); //calc distance//

			if (dist <= searchRad * searchRad)
				pcount++;
			
		}

		sncount[qspxl] = pcount;

	}

}

void CopyCountsCuda(int qnum, int len, float searchRad, int globalW, int globalH, float* pvmMat, float* vpos, int* xfcount, int* xfoffset, int* FragVertex, bool* pixelIn, int* sncount)
{
	CopyCountsKernel << < (qnum * len * len) / 256 + 1, 256 >> > (qnum, len, searchRad, globalW, globalH, pvmMat, vpos, xfcount, xfoffset, FragVertex, pixelIn, sncount);

}

void CreateNbsOffsetArrayCuda(int n, int* sncount, int* snoffset)
{
	thrust::device_ptr<int> o = thrust::device_pointer_cast(snoffset);
	thrust::device_ptr<int> c = thrust::device_pointer_cast(sncount);

	//call thrust function
	thrust::exclusive_scan(c, c + n, o);
}


int SumNbsCuda(int n, int* sncount)
{

	thrust::device_ptr<int> c = thrust::device_pointer_cast(sncount);

	//get count of xfcount//
	int NbsNum = thrust::reduce(c, c + n, (int)0, thrust::plus<int>());

	return NbsNum;
}

__device__
unsigned long long GenerateVertexDistKey(int vertex, float dist)
{
	unsigned long long result = vertex;
	result = result << 32;

	//unsigned long long result=0;

	const float lineParameter = dist;
	//uint converted_key = *((uint *)&lineParameter);
	unsigned int converted_key = *((unsigned int*)&lineParameter);
	const unsigned int mask = ((converted_key & 0x80000000) ? 0xffffffff : 0x80000000);
	converted_key ^= mask;

	result |= (unsigned long long)(converted_key);

	return result;

}


__global__
void FillDistanceKernel(int qnum, int len, float searchRad, int globalW, int globalH, float* pvmMat, float* vpos, int* xfcount, int* xfoffset, int* FragVertex, bool* pixelIn, int* sncount, int* snoffset, int* NbVertex, unsigned long long* NbVertexDist)
{
	int qspxl = blockIdx.x * blockDim.x + threadIdx.x;

	if (qspxl < qnum * len * len)
	{
		int q = qspxl / (len * len);
		int spxl = qspxl % (len * len);
		int sx = spxl % len;
		int sy = spxl / len;

		if (!pixelIn[spxl])
			return;

		//get vertex//
		float qx = vpos[3 * q + 0];
		float qy = vpos[3 * q + 1];
		float qz = vpos[3 * q + 2];
		float qw = 1.0;

		float posXpvm = pvmMat[0] * qx + pvmMat[4] * qy + pvmMat[8] * qz + pvmMat[12] * qw;
		float posYpvm = pvmMat[1] * qx + pvmMat[5] * qy + pvmMat[9] * qz + pvmMat[13] * qw;
		float posZpvm = pvmMat[2] * qx + pvmMat[6] * qy + pvmMat[10] * qz + pvmMat[14] * qw;
		float posWpvm = pvmMat[3] * qx + pvmMat[7] * qy + pvmMat[11] * qz + pvmMat[15] * qw;

		//exact pixel of q//
		int qxscreen = (int)(((posXpvm / posWpvm) / 2 + 0.5) * globalW);
		int qyscreen = (int)(((posYpvm / posWpvm) / 2 + 0.5) * globalH);

		//pixel in the square, based on thread//
		int xscreen = (qxscreen - len/2) + sx;
		int yscreen = (qyscreen - len/2) + sy;
		int pxl = xscreen + yscreen * globalW;

		if (xscreen<0 || xscreen>globalW - 1 || yscreen<0 || yscreen>globalH - 1)
			return;

		if (xfcount[pxl] == 0)
			return;

		int offset = xfoffset[pxl];
		for (int v = 0; v < xfcount[pxl]; v++)
		{
			int vx = FragVertex[v + offset];
			float x = vpos[3 * vx + 0];
			float y = vpos[3 * vx + 1];
			float z = vpos[3 * vx + 2];

			float dist = (x - qx) * (x - qx) + (y - qy) * (y - qy) + (z - qz) * (z - qz);

			if (dist <= searchRad * searchRad)
			{
				int pos = atomicAdd(&sncount[qspxl], 1);

				NbVertex[snoffset[qspxl] + pos] = vx;
				NbVertexDist[snoffset[qspxl] + pos] = GenerateVertexDistKey(q, dist);
			}
			


		}

	}

}

void FillDistanceCuda(int qnum, int len, float searchRad, int globalW, int globalH, float* pvmMat, float* vpos, int* xfcount, int* xfoffset, int* FragVertex, bool* pixelIn, int* sncount, int* snoffset, int* NbVertex, unsigned long long* NbVertexDist)
{

	FillDistanceKernel << < (qnum * len * len) / 256 + 1, 256 >> > (qnum, len, searchRad, globalW, globalH, pvmMat, vpos, xfcount, xfoffset, FragVertex, pixelIn, sncount, snoffset, NbVertex, NbVertexDist);


}

void SortNeighborsCuda(int NbsNum, int* NbVertex, unsigned long long* NbVertexDist)
{

	//device pointers//
	thrust::device_ptr<int> fv = thrust::device_pointer_cast(NbVertex);
	thrust::device_ptr<unsigned long long> fvd = thrust::device_pointer_cast(NbVertexDist);

	//tmp buffers for thrust::gather//
	int* NbVertexTmp;
	hipMalloc((void**)&NbVertexTmp, NbsNum * sizeof(int));
	thrust::device_ptr<int> fvt = thrust::device_pointer_cast(NbVertexTmp);

	//init an index buffer//
	unsigned int* NbIndex;
	hipMalloc((void**)&NbIndex, NbsNum * sizeof(unsigned int));
	thrust::device_ptr<unsigned int> fi = thrust::device_pointer_cast(NbIndex);
	thrust::sequence(fi, fi + NbsNum, 0);


	//sort depth and index//
	thrust::sort_by_key(fvd, fvd + NbsNum, fi);


	//change all other arrays based on the sorted index//
	thrust::gather(fi, fi + NbsNum, fv, fvt);
	hipMemcpy(NbVertex, NbVertexTmp, NbsNum * sizeof(int), hipMemcpyDeviceToDevice);

}

void CopyKNeighborsCuda(int k, float SearchRad, int qnum, int len, int* sncount, int NbsNum, int* NbVertex, int vnum, float* vpos, vector<vector<int>>& Nbs)
{
	int* NbVertexHost = new int[NbsNum];
	hipMemcpy(NbVertexHost, NbVertex, NbsNum * sizeof(int), hipMemcpyDeviceToHost);
	int* sncountHost = new int[qnum * len * len];
	hipMemcpy(sncountHost, sncount, qnum * len * len * sizeof(int), hipMemcpyDeviceToHost);
	float* vposHost = new float[vnum * 3];
	hipMemcpy(vposHost, vpos, vnum * 3 * sizeof(float), hipMemcpyDeviceToHost);

	int offset = 0;
	for (int q = 0; q < qnum; q++)
	{
		int start = len * len * q;
		int end = start + len * len;
		int acc = 0;
		for (int i = start; i < end; i++)
		{
			acc += sncountHost[i];
		}
		
		int copyn;
		if (acc < k)
			copyn = acc;
		else
			copyn = k;

		int counter = 0;
		for (int i = 0; i < copyn; i++)
		{
			int vx = NbVertexHost[offset + i];


			float dist = sqrt((vposHost[3 * q + 0] - vposHost[3 * vx + 0]) * (vposHost[3 * q + 0] - vposHost[3 * vx + 0]) + (vposHost[3 * q + 1] - vposHost[3 * vx + 1]) * (vposHost[3 * q + 1] - vposHost[3 * vx + 1]) + (vposHost[3 * q + 2] - vposHost[3 * vx + 2]) * (vposHost[3 * q + 2] - vposHost[3 * vx + 2]));

			if (dist < SearchRad)
				Nbs[q][counter++] = vx;
		}
		offset += acc;

	}

}


